#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdio.h>

// Error handling macro
#define CHECK_CUDNN(expression)                                                 \
  do {                                                                          \
    hipdnnStatus_t status = (expression);                                        \
    if (status != HIPDNN_STATUS_SUCCESS) {                                       \
      fprintf(stderr, "Error at line %d: %s\n", __LINE__, hipdnnGetErrorString(status)); \
      exit(EXIT_FAILURE);                                                       \
    }                                                                           \
  } while (0)

// Function for convolution using CuDNN
void cudnnConvolution(
    float *inputImage, 
    float *kernel, 
    int imageSize, 
    int kernelSize, 
    float *outputImage
) {
  // CuDNN handle
  hipdnnHandle_t cudnnHandle;
  CHECK_CUDNN(hipdnnCreate(&cudnnHandle));

  // Data tensor descriptors
  hipdnnTensorDescriptor_t inputDescriptor, kernelDescriptor, outputDescriptor;
  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&inputDescriptor));
  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&kernelDescriptor));
  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&outputDescriptor));

  // Set tensor dimensions (assuming NCHW format)
  const int n = 1, c = 1, h = imageSize, w = imageSize;
  const int k = 1, c_k = 1, h_k = kernelSize, w_k = kernelSize;
  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(inputDescriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));
  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(kernelDescriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, k, c_k, h_k, w_k));
  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(outputDescriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));

  // Convolution descriptor
  hipdnnConvolutionDescriptor_t convolutionDescriptor;
  CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&convolutionDescriptor));
  CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(convolutionDescriptor, 0, 0, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT));

  // Set up convolution algorithm
  hipdnnConvolutionFwdAlgo_t convolutionAlgorithm;
  CHECK_CUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle, inputDescriptor, kernelDescriptor, 
                                                 convolutionDescriptor, outputDescriptor, 
                                                 HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &convolutionAlgorithm));

  // Workspace size and allocation
  size_t workspaceSizeInBytes;
  CHECK_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle, inputDescriptor, kernelDescriptor, 
                                                   convolutionDescriptor, outputDescriptor, 
                                                   convolutionAlgorithm, &workspaceSizeInBytes));
  void *workspace = nullptr;
  if (workspaceSizeInBytes > 0) {
    CHECK_CUDNN(hipMalloc(&workspace, workspaceSizeInBytes));
  }

  // Perform convolution
  const float alpha = 1.0f, beta = 0.0f;
  CHECK_CUDNN(hipdnnConvolutionForward(cudnnHandle, &alpha, inputDescriptor, inputImage,
                                     kernelDescriptor, kernel, convolutionDescriptor, 
                                     convolutionAlgorithm, workspace, workspaceSizeInBytes,
                                     &beta, outputDescriptor, outputImage));

  // Cleanup
  if (workspace) {
    hipFree(workspace);
  }
  hipdnnDestroyTensorDescriptor(inputDescriptor);
  hipdnnDestroyTensorDescriptor(kernelDescriptor);
  hipdnnDestroyTensorDescriptor(outputDescriptor);
  hipdnnDestroyConvolutionDescriptor(convolutionDescriptor);
  hipdnnDestroy(cudnnHandle);
}
// CONVOLUTION CUDA
double A1(DATA_TYPE *IMG_IN, DATA_TYPE *KERNEL_IN_H, const int IMAGE_SIZE, const int KERNEL_SIZE)
{
	// DEFINING NEW GPU MEMORY POINTERS
	DATA_TYPE *IMG_IN_D;
	DATA_TYPE *IMG_OUT_D;
	const int SIZE_IN_BYTES = sizeof(DATA_TYPE) * IMAGE_SIZE * IMAGE_SIZE;

	// ALLOCATING GPU MEMORY
	hipMalloc((void **)&IMG_IN_D, SIZE_IN_BYTES);
	hipMalloc((void **)&IMG_OUT_D, SIZE_IN_BYTES);

	// COPYING PROBLEM VARIABLE TO GPU
	hipMemcpy(IMG_IN_D, IMG_IN_H, SIZE_IN_BYTES, hipMemcpyHostToDevice);

	// DEFINING DIMENSIONS OF THE KERNEL EXECUTION
    dim3 block(THREAD_BLOCK_DIM_X, THREAD_BLOCK_DIM_Y);
	dim3 grid(
		(size_t) ceil( ((float) IMAGE_SIZE) / ((float) block.x) ), 
		(size_t) ceil( ((float) IMAGE_SIZE) / ((float) block.y) ) 
	);

	// MEASURING THE EXECUTION TIME
	clock_t t;
	t = clock();

	// KERNEL INVOCATION
	conv2Dkernel_base<<<grid, block>>>(IMG_IN_D, IMG_OUT_D, IMAGE_SIZE);
	hipDeviceSynchronize();

	// MEASURING AND DISPLAYING EXECUTION TIMR
	t = clock() - t;
	double time_taken_in_seconds = ((double)t) / CLOCKS_PER_SEC;

    // CUDA ERROR DETECTION
    hipError_t err = hipGetLastError();
	if (err != hipSuccess && false){
		printf("CUDA Error - %s : %s\n",hipGetErrorName(err), hipGetErrorString(err));
	}

    // DISPLAYING RESULTS
	// printf("CUDA Runtime: %f\n", time_taken_in_seconds);

	// COPYING RESULT VARIABL TO HOST
	hipMemcpy(IMG_OUT_H, IMG_OUT_D, SIZE_IN_BYTES, hipMemcpyDeviceToHost);

	// FREEING GPU MEMORY
	hipFree(IMG_IN_D);
	hipFree(IMG_OUT_D);

	return time_taken_in_seconds;
}