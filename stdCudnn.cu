#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define CHECK_CUDNN(status) \
    if (status != HIPDNN_STATUS_SUCCESS) { \
        printf("Error: %s\n", hipdnnGetErrorString(status)); \
        exit(EXIT_FAILURE); \
    }

int myStdCudnn(float* IMG_IN, float* IMG_OUT, float* FILTER_IN, int IMAGE_SIZE, int FILTER_SIZE) {
    // Initialize cuDNN
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // Define image and filter dimensions
    int IMAGE_SIZE = IMAGE_SIZE;  // Size of input image (assuming square image)
    int FILTER_SIZE = FILTER_SIZE;  // Size of filter (assuming square filter)
    int imgBytes = IMAGE_SIZE * IMAGE_SIZE * sizeof(float);
    int filterBytes = FILTER_SIZE * FILTER_SIZE * sizeof(float);

    // Allocate memory for input image, filter, and output
    // float *h_input = (float *) malloc(imgBytes);
    float *h_output = (float *) malloc(imgBytes);

    // float *FILTER_IN = (float *)malloc(filterBytes);
    // float FILTER_IN[] = {2,0,0, 0,0,0, 0,0,0};

    // Initialize input image and filter
    initializeImage(h_input, IMAGE_SIZE * IMAGE_SIZE);
    // initializeImage(FILTER_IN, FILTER_SIZE * FILTER_SIZE);

    // Allocate device memory for input image, filter, and output
    float *d_input, *d_filter, *d_output;
    hipMalloc(&d_input, imgBytes);
    hipMalloc(&d_filter, filterBytes);
    hipMalloc(&d_output, imgBytes);

    // Copy input image and filter to device
    hipMemcpy(d_input, h_input, imgBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_filter, FILTER_IN, filterBytes, hipMemcpyHostToDevice);

    // Define convolution parameters
    hipdnnConvolutionDescriptor_t convDesc;
    hipdnnCreateConvolutionDescriptor(&convDesc);
    int pad = 0;  // Padding
    int stride = 1;  // Stride
    int dilation = 1;  // Dilation
    hipdnnSetConvolution2dDescriptor(convDesc, pad, pad, stride, stride, dilation, dilation, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

    // Define convolution operation parameters
    hipdnnTensorDescriptor_t inputDesc, outputDesc;
    hipdnnCreateTensorDescriptor(&inputDesc);
    hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, IMAGE_SIZE, IMAGE_SIZE);
    hipdnnCreateTensorDescriptor(&outputDesc);
    hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, IMAGE_SIZE - FILTER_SIZE + 1, IMAGE_SIZE - FILTER_SIZE + 1);
    hipdnnFilterDescriptor_t filterDesc;
    hipdnnCreateFilterDescriptor(&filterDesc);
    hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 1, 1, FILTER_SIZE, FILTER_SIZE);

    // Perform convolution
    hipdnnConvolutionFwdAlgo_t algo;
    // cudnnGetConvolutionForwardAlgorithm_v7(cudnn, inputDesc, filterDesc, convDesc, outputDesc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM, 0, &algo);
    hipdnnConvolutionFwdAlgoPerf_t perfResults;
    int numAlgos;

    clock_t t;
    t = clock();

    cudnnGetConvolutionForwardAlgorithm_v7(cudnn, inputDesc, filterDesc, convDesc, outputDesc, 1, &numAlgos, &perfResults);
    algo = perfResults.algo;
    printf("%d",algo);

    // t = clock() - t;
    // double time_taken_in_seconds = ((double)t) / CLOCKS_PER_SEC;


    void *workSpace = NULL;
    size_t workSpaceSize = 0;
    hipdnnGetConvolutionForwardWorkspaceSize(cudnn, inputDesc, filterDesc, convDesc, outputDesc, algo, &workSpaceSize);
    if (workSpaceSize > 0) {
        hipMalloc(&workSpace, workSpaceSize);
    }
    float alpha = 1.0f;
    float beta = 0.0f;
    hipdnnConvolutionForward(cudnn, &alpha, inputDesc, d_input, filterDesc, d_filter, convDesc, algo, workSpace, workSpaceSize, &beta, outputDesc, d_output);

    t = clock() - t;
    double time_taken_in_seconds = ((double)t) / CLOCKS_PER_SEC;

    // Copy output from device to host
    hipMemcpy(h_output, d_output, imgBytes, hipMemcpyDeviceToHost);

    printf("Time %f \n",time_taken_in_seconds * 1000);
    // Print output
    printf("Output image:\n");
    for (int i = 0; i < IMAGE_SIZE - FILTER_SIZE + 1; ++i) {
        for (int j = 0; j < IMAGE_SIZE - FILTER_SIZE + 1; ++j) {
            printf("%f ", h_output[i * (IMAGE_SIZE - FILTER_SIZE + 1) + j]);
        }
        printf("\n");
    }
    printf("\n");
    for (int i = 0; i < FILTER_SIZE; ++i) {
        for (int j = 0; j < FILTER_SIZE; ++j) {
            printf("%f ", FILTER_IN[i * FILTER_SIZE + j]);
        }
        printf("\n");
    }
    printf("\n");
    for (int i = 0; i < IMAGE_SIZE; ++i) {
        for (int j = 0; j < IMAGE_SIZE; ++j) {
            printf("%f ", h_input[i * IMAGE_SIZE + j]);
        }
        printf("\n");
    }

    // Cleanup
    // free(h_input);
    // free(FILTER_IN);
    free(h_output);
    hipFree(d_input);
    hipFree(d_filter);
    hipFree(d_output);
    hipdnnDestroyTensorDescriptor(inputDesc);
    hipdnnDestroyTensorDescriptor(outputDesc);
    hipdnnDestroyFilterDescriptor(filterDesc);
    hipdnnDestroyConvolutionDescriptor(convDesc);
    hipdnnDestroy(cudnn);

    return 0;
}
