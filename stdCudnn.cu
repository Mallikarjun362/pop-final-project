#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define CHECK_CUDNN(status) \
    if (status != HIPDNN_STATUS_SUCCESS) { \
        printf("Error: %s\n", hipdnnGetErrorString(status)); \
        exit(EXIT_FAILURE); \
    }

void initializeImage(float *img, int size) {
    for (int i = 0; i < size; ++i) {
        img[i] = (float)rand() / RAND_MAX;
    }
}

int main() {
    // Initialize cuDNN
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // Define image and filter dimensions
    int imgSize = 4;  // Size of input image (assuming square image)
    int filterSize = 3;  // Size of filter (assuming square filter)
    int imgBytes = imgSize * imgSize * sizeof(float);
    int filterBytes = filterSize * filterSize * sizeof(float);

    // Allocate memory for input image, filter, and output
    float *h_input = (float *)malloc(imgBytes);
    float *h_filter = (float *)malloc(filterBytes);
    float *h_output = (float *)malloc(imgBytes);

    // Initialize input image and filter
    initializeImage(h_input, imgSize * imgSize);
    initializeImage(h_filter, filterSize * filterSize);

    // Allocate device memory for input image, filter, and output
    float *d_input, *d_filter, *d_output;
    hipMalloc(&d_input, imgBytes);
    hipMalloc(&d_filter, filterBytes);
    hipMalloc(&d_output, imgBytes);

    // Copy input image and filter to device
    hipMemcpy(d_input, h_input, imgBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_filter, h_filter, filterBytes, hipMemcpyHostToDevice);

    // Define convolution parameters
    hipdnnConvolutionDescriptor_t convDesc;
    hipdnnCreateConvolutionDescriptor(&convDesc);
    int pad = 0;  // Padding
    int stride = 1;  // Stride
    int dilation = 1;  // Dilation
    hipdnnSetConvolution2dDescriptor(convDesc, pad, pad, stride, stride, dilation, dilation, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

    // Define convolution operation parameters
    hipdnnTensorDescriptor_t inputDesc, outputDesc;
    hipdnnCreateTensorDescriptor(&inputDesc);
    hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, imgSize, imgSize);
    hipdnnCreateTensorDescriptor(&outputDesc);
    hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, imgSize - filterSize + 1, imgSize - filterSize + 1);
    hipdnnFilterDescriptor_t filterDesc;
    hipdnnCreateFilterDescriptor(&filterDesc);
    hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 1, 1, filterSize, filterSize);

    // Define convolution algorithm
    hipdnnConvolutionFwdAlgo_t algo;
    hipdnnGetConvolutionForwardAlgorithm(cudnn, inputDesc, filterDesc, convDesc, outputDesc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo);

    // Perform convolution
    void *workSpace = NULL;
    size_t workSpaceSize = 0;
    hipdnnStatus_t status = hipdnnGetConvolutionForwardWorkspaceSize(cudnn, inputDesc, filterDesc, convDesc, outputDesc, algo, &workSpaceSize);
    CHECK_CUDNN(status);
    if (workSpaceSize > 0) {
        hipMalloc(&workSpace, workSpaceSize);
    }
    float alpha = 1.0f;
    float beta = 0.0f;
    status = hipdnnConvolutionForward(cudnn, &alpha, inputDesc, d_input, filterDesc, d_filter, convDesc, algo, workSpace, workSpaceSize, &beta, outputDesc, d_output);
    CHECK_CUDNN(status);

    // Copy output from device to host
    hipMemcpy(h_output, d_output, imgBytes, hipMemcpyDeviceToHost);

    // Print output
    printf("Output image:\n");
    for (int i = 0; i < imgSize - filterSize + 1; ++i) {
        for (int j = 0; j < imgSize - filterSize + 1; ++j) {
            printf("%f ", h_output[i * (imgSize - filterSize + 1) + j]);
        }
        printf("\n");
    }

    // Cleanup
    free(h_input);
    free(h_filter);
    free(h_output);
    hipFree(d_input);
    hipFree(d_filter);
    hipFree(d_output);
    hipdnnDestroyTensorDescriptor(inputDesc);
    hipdnnDestroyTensorDescriptor(outputDesc);
    hipdnnDestroyFilterDescriptor(filterDesc);
    hipdnnDestroyConvolutionDescriptor(convDesc);
    hipdnnDestroy(cudnn);

    return 0;
}
