#include "hip/hip_runtime.h"
// CONVOLUTION CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void convolution_kernel(
    const float* input, const float* filter, float* output,
    int input_width, int input_height, int filter_width, int filter_height
) {
  // Thread ID and block dimensions
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int threads_per_block = blockDim.x;

  // Calculate output element coordinates
  int output_x = bid * threads_per_block + tid;
  int output_y = blockIdx.y;

  // Check if within output bounds
  if (output_x >= input_width || output_y >= input_height) return;

  // Calculate padding and stride (assuming padding = 1, stride = 1)
  int padding = (filter_width - 1) / 2;

  // Temporary buffer for input elements
  float iTemp[filter_width];

  // Load first and last elements for each thread
  iTemp[0] = input[(output_y * input_width) + output_x];
  iTemp[filter_width - 1] = input[(output_y * input_width) + output_x + filter_width - 1];

  // Apply column reuse optimization (Algorithm 1)
  if (tid < threads_per_block - 2) {
    unsigned long long exchange;
    asm volatile ("mov.b64 %0, {%1, %2};" : "=l"(exchange) : "r"(iTemp[0]), "r"(iTemp[filter_width - 1]));
    int shift = ((tid + 2) & 2) << 4;
    asm volatile ("shr.b64 %0, %1, %2;" : "=l"(exchange) : "r"(exchange), "r"(shift));
    asm volatile ("mov.b64 {%0, %1}, %2;" : "=r"(iTemp[1]), "=r"(iTemp[2]) : "l"(exchange));
    iTemp[2] = __shfl_xor(iTemp[1], 2);
  }
  __syncthreads();

  // Perform convolution for the current output element
  float sum = 0.0f;
  for (int fy = 0; fy < filter_height; fy++) {
    for (int fx = 0; fx < filter_width; fx++) {
      int input_x = output_x + fx - padding;
      int input_y = output_y + fy - padding;

      // Check if within input bounds (ignoring padding)
      if (input_x >= 0 && input_x < input_width && input_y >= 0 && input_y < input_height) {
        sum += iTemp[fx] * filter[(fy * filter_width) + fx];
      }
    }
  }
  output[(output_y * input_width) + output_x] = sum;
}

// Host code to launch kernel
double A2ColumnReuse(float* IMG_IN, float* IMG_OUT, float*  FILTER_IN, int IMAGE_SIZE, int FILTER_SIZE)
{
    int imgBytes = IMAGE_SIZE * IMAGE_SIZE * sizeof(float);
    int filterBytes = FILTER_SIZE * FILTER_SIZE * sizeof(float);

    float *d_input, *d_filter, *d_output;
    hipMalloc(&d_input, imgBytes);
    hipMalloc(&d_filter, filterBytes);
    hipMalloc(&d_output, imgBytes);

    hipMemcpy(d_input, IMG_IN, imgBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_filter, FILTER_IN, filterBytes, hipMemcpyHostToDevice);

    // Define block and grid sizes
    int threads_per_block = 32;
    int blocks_per_grid_x = (IMAGE_SIZE + threads_per_block - 1) / threads_per_block;
    int blocks_per_grid_y = IMAGE_SIZE;

    dim3 block_size(threads_per_block);
    dim3 grid_size(blocks_per_grid_x, blocks_per_grid_y);
    // Launch kernel
    clock_t t;
    t = clock();
    
    convolution_kernel<<<grid_size, block_size>>>(d_input, d_filter, d_output, IMAGE_SIZE, IMAGE_SIZE, FILTER_SIZE, FILTER_SIZE);
    hipDeviceSynchronize();

    t = clock() - t;
    double time_taken_in_seconds = ((double)t) / CLOCKS_PER_SEC;

    return time_taken_in_seconds * 1000
}